#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <stdio.h>
#include <stdlib.h>
using namespace std;
#include <iterator>
#include <algorithm>
#include <random>
#include <math.h>


__constant__ float gpu_iones[12000];
// Cada thread deberia calcular la carga de 1 punto
__global__ void calcular_carga(float* cargas, int cantidad) {
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    
	if(tId < 8192*8192) {
        
        float x = tId%8192;
        float y = tId/8192;
        
        float carga = 0;
        float distancia;
        float x_2, y_2;
        for (int i = 0; i < cantidad; i++)  {
            x_2 = (x - gpu_iones[i]) * (x - gpu_iones[i]);
            y_2 = (y - gpu_iones[6000+i]) * (y - gpu_iones[6000+i]);
            distancia = sqrt(x_2 + y_2);
            if (distancia != 0)  {
                carga += 1.0 / distancia;
            } else {
                carga+=1;
            }
            
        }
    
        cargas[tId] = carga;
    }
    
}

// cada thread calcula la menor carga en su fila y la guarda

__global__ void calcular_carga_fila(float* cargas, float*cargas_menores, int cantidad) {
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    

    if(tId < 8192) {
        float Q_menor = cargas[tId*8192];
        float y = tId;
        float x;
        

        for (int i = tId*8192; i < tId*8192+8192; i++)  {
            if(cargas[i] <Q_menor){
                Q_menor = cargas[i];
                x = i%8192;
            }
        }
        cargas_menores[tId*3] = Q_menor;
        cargas_menores[tId*3+1] = x;
        cargas_menores[tId*3+2] = y;
    }
    
}
// Calculamos entre todas la menor y ponemos la carga ahí
__global__ void posicionar_ion(float*cargas_menores, int cantidad, float* salida_gpu) {
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    

    if(tId < 1) {
        float Q_menor = cargas_menores[0];
        float x = cargas_menores[1];
        float y = cargas_menores[2];

        for (int i = 0; i < 8192*3; i+=3)  {
            
            if(cargas_menores[i] < Q_menor){
                 
                
                Q_menor = cargas_menores[i];
                
                x = cargas_menores[i+1];
                y = cargas_menores[i+2];
               
                
            }
           
        }
        salida_gpu[0] = x;
        salida_gpu[1] = y; 
    }

    
    
}



int main(int argc, char const *argv[])
{
    
    float *gpu_cargas, *cargas_menores, *iones, *salida, *salida_gpu;
    hipEvent_t ct1, ct2;
    float dt;
    int cantidad;
    iones = new float[12000];
    salida = new float[2];

    int block_size = 256;
    int grid_size = (int) ceil( (float) 8192*8192 / block_size);
    int grid_size_b = (int) ceil( (float) 8192 / block_size);
    int grid_size_c = (int) ceil( (float) 1 / block_size);

    
    FILE *in = fopen("dataset", "r");
    for (int i = 0; i < 5000; i++)
    {
        fscanf(in, "%f %f", &iones[i], &iones[6000+i]);
       // cout << iones[i] << " " << iones[6000+i] << endl;
    }



    hipMalloc(&gpu_cargas, sizeof(float) * 8192 * 8192);
    hipMalloc(&cargas_menores, sizeof(float) * 8192*3);
    hipMalloc(&salida_gpu, sizeof(float) *2);

    

    hipMemcpyToSymbol(HIP_SYMBOL(gpu_iones), iones,sizeof(float) * 12000, 0, hipMemcpyHostToDevice);
 
    hipEventCreate(&ct1);
	hipEventCreate(&ct2);
    hipEventRecord(ct1);

    
    for (cantidad = 5000; cantidad < 5100; cantidad++)
    {
        
        
        calcular_carga<<<grid_size, block_size>>>(gpu_cargas, cantidad);
        hipDeviceSynchronize();

        calcular_carga_fila<<<grid_size_b, block_size>>>(gpu_cargas, cargas_menores, cantidad);
        hipDeviceSynchronize();

        posicionar_ion<<<grid_size_c, block_size>>>(cargas_menores, cantidad, salida_gpu);
        hipDeviceSynchronize();
        hipMemcpy(salida, salida_gpu,sizeof(float) * 2, hipMemcpyDeviceToHost);

        cout << salida[0] << " " << salida[1] << endl;

        iones[cantidad] = salida[0];
        iones[cantidad+6000] = salida[1];

        hipMemcpyToSymbol(HIP_SYMBOL(gpu_iones), iones,sizeof(float) * 12000, 0, hipMemcpyHostToDevice);
    }

    hipEventRecord(ct2);
	hipEventSynchronize(ct2);
    hipEventElapsedTime(&dt, ct1, ct2);

    cout << "Tiempo: " << dt << "[ms]" << '\n';
    

    hipFree(gpu_cargas);
    hipFree(cargas_menores);
    hipFree(salida_gpu);

    close(in);

    delete iones;
    delete salida;
    
    return 0;
}