
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <stdio.h>
#include <stdlib.h>
using namespace std;
#include <iterator>
#include <algorithm>
#include <random>
#include <math.h>

// Cada thread deberia calcular la carga de 1 punto
__global__ void calcular_carga(float* iones_x, float* iones_y, float* cargas, int cantidad) {
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    
	if(tId < 8192*8192) {
        
        int x = tId/8192;
        int y = tId%8192;
        
        float carga = 0;
        float distancia;
        float x_2, y_2;
        carga = 0;
        for (int i = 0; i < cantidad; i++)  {
            x_2 = (x - iones_x[i]) * (x - iones_x[i]);
            y_2 = (y - iones_y[i]) * (y - iones_y[i]);
            distancia = sqrt(x_2 + y_2);
            if (distancia == 0)  {
                distancia = 0.0000000000001;
            }
            carga += 1.0 / distancia;
        }
    
        cargas[tId] = carga;
    }
    
}

// cada thread calcula la menor carga en su fila y la guarda

__global__ void calcular_carga_fila(float* iones_x, float* iones_y, float* cargas, float*cargas_menores, int cantidad) {
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    

    if(tId < 8192) {
        float Q_menor = 100000000000;
        int x = tId;
        int y;
        

        for (int i = tId*8192; i < tId*8192+8192; i++)  {
            if(cargas[i] <Q_menor){
                Q_menor = cargas[i];
                y = i%8192;
            }
        }
        cargas_menores[tId*3] = Q_menor;
        cargas_menores[tId*3+1] = x;
        cargas_menores[tId*3+2] = y;
    }
    
}
// Calculamos entre todas la menor y ponemos la carga ahí
__global__ void posicionar_ion(float* iones_x, float* iones_y, float*cargas_menores, int cantidad) {
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    float Q_menor = 100000000000;
    int a;
    int b;

    if(tId < 1) {
        for (int i = 0; i < 8192*3; i+=3)  {
            if(cargas_menores[i] < Q_menor){
                printf("%i %f \n", i, Q_menor);
                Q_menor = cargas_menores[i];
                
                a = cargas_menores[i+1];
                b = cargas_menores[i+2];
                
            }
        }
        iones_x[cantidad] = a;
        iones_y[cantidad] = b; 
    }

    
    
}



int main(int argc, char const *argv[])
{
    
    float *gpu_cargas, *cargas_menores, *cargas, *gpu_iones_x, *gpu_iones_y, *iones_x, *iones_y;
    hipEvent_t ct1, ct2;
    float dt;
    int cantidad;
    iones_x = new float[6000];
    iones_y = new float[6000];
    cargas = new float[8192*8192];

    int block_size = 256;
    int grid_size = (int) ceil( (float) 8192*8182 / block_size);
    int grid_size_b = (int) ceil( (float) 8192 / block_size);
    int grid_size_c = (int) ceil( (float) 1 / block_size);

    
    FILE *in = fopen("dataset", "r");
    for (int i = 0; i < 5000; i++)
    {
        fscanf(in, "%f %f", &iones_x[i], &iones_y[i]);
        //cout << iones_x[i] << " " << iones_y[i] << endl;
    }


    hipMalloc(&gpu_iones_x, sizeof(float) * 6000);
    hipMalloc(&gpu_iones_y, sizeof(float) * 6000);
    hipMalloc(&gpu_cargas, sizeof(float) * 8192 * 8192);
    hipMalloc(&cargas_menores, sizeof(float) * 8192*3);

    hipMemcpy(gpu_cargas, cargas, sizeof(float) * 8192 * 8192, hipMemcpyHostToDevice);
    

    hipEventCreate(&ct1);
	hipEventCreate(&ct2);
    hipEventRecord(ct1);

    
    for (cantidad = 5000; cantidad < 5100; cantidad++)
    {
        hipMemcpy(gpu_iones_x, iones_x, sizeof(float) * 6000, hipMemcpyHostToDevice);
        hipMemcpy(gpu_iones_y, iones_y, sizeof(float) * 6000, hipMemcpyHostToDevice);
        calcular_carga<<<grid_size, block_size>>>(gpu_iones_x, gpu_iones_y, gpu_cargas, cantidad);
        hipDeviceSynchronize();

        

        hipDeviceSynchronize();
        //cout << "Calculando carga fila para " <<  cantidad << endl;
        calcular_carga_fila<<<grid_size_b, block_size>>>(gpu_iones_x, gpu_iones_y, gpu_cargas, cargas_menores, cantidad);
        hipDeviceSynchronize();
        //cout << "Posicionado ion para " <<  cantidad << endl;
        posicionar_ion<<<grid_size_c, block_size>>>(gpu_iones_x, gpu_iones_y, cargas_menores, cantidad);
        hipDeviceSynchronize();
        hipMemcpy(iones_x, gpu_iones_x,sizeof(float) * 6000, hipMemcpyDeviceToHost);
        hipMemcpy(iones_y, gpu_iones_y,sizeof(float) * 6000, hipMemcpyDeviceToHost);
        cout << iones_x[cantidad] << " " << iones_y[cantidad] << endl;
    }

    hipEventRecord(ct2);
	hipEventSynchronize(ct2);
    hipEventElapsedTime(&dt, ct1, ct2);

    cout << "Tiempo: " << dt << "[ms]" << '\n';

    hipFree(gpu_iones_x);
    hipFree(gpu_iones_y);
    hipFree(cargas);
    hipFree(cargas_menores);

    delete iones_x;
    delete iones_y;
    
    return 0;
}