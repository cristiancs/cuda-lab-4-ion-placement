
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <stdio.h>
#include <stdlib.h>
using namespace std;
#include <iterator>
#include <algorithm>
#include <random>
#include <math.h>

// Cada thread deberia calcular la carga de 1 punto
__global__ void calcular_carga(float* iones_x, float* iones_y, float* cargas, int cantidad) {
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    
	if(tId < 8192*8192) {
        int a = tId/8192;
        int b = tId%8192;
       
        float carga = 0;
        float distancia;
        float x_2, y_2;
        carga = 0;
        for (int i = 0; i < cantidad; i++)  {
            x_2 = (a - iones_x[i]) * (a - iones_x[i]);
            y_2 = (b - iones_y[i]) * (b - iones_y[i]);
            distancia = sqrt(x_2 + y_2);
            if (distancia == 0)  {
                distancia = 0.0000000000001;
            }
            carga += 1.0 / distancia;
        }
        cargas[tId] = carga;
    }
    
}

// cada thread calcula la menor carga en su fila y la guarda

__global__ void calcular_carga_fila(float* iones_x, float* iones_y, float* cargas, float*cargas_menores, int cantidad) {
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    float Q_menor = cargas[tId*8192];
    int a = tId;
    int b;

    if(tId < 8192) {
        for (int i = tId*8192; i < tId*8192+8192; i++)  {
            if(cargas[i] <Q_menor){
                Q_menor = cargas[i];
                b = i;
            }
        }
        cargas_menores[tId*3] = Q_menor;
        cargas_menores[tId*3+1] = a;
        cargas_menores[tId*3+2] = b;
    }
    
}
// Calculamos entre todas la menor y ponemos la carga ahí
__global__ void posicionar_ion(float* iones_x, float* iones_y, float*cargas_menores, int cantidad) {
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    float Q_menor = cargas_menores[0];
    int a = tId;
    int b;

    if(tId < 8192) {
        for (int i = tId*3; i < 8192*3; i++)  {
            if(cargas_menores[i] < Q_menor){
                Q_menor = cargas_menores[i];
                a = cargas_menores[i+1];
                b = cargas_menores[i+2];
            }
        }
    }
    iones_x[cantidad] = a;
    iones_y[cantidad] = b; 
    
}



int main(int argc, char const *argv[])
{
    float *iones_x, *iones_y;
    float *gpu_iones_x, *gpu_iones_y, *cargas, *cargas_menores;
    float x, y;
    hipEvent_t ct1, ct2;
    float dt;
    int cantidad;

    iones_x = new float[6000];
    iones_y = new float[6000];

    int block_size = 256;
    int grid_size = (int) ceil( (float) 8192*8182 / block_size);
    int grid_size_b = (int) ceil( (float) 8192 / block_size);

    std::random_device rd;
    std::default_random_engine generator(rd()); // rd() provides a random seed
    std::uniform_real_distribution<double> distribution(0.1, 8192);

    for (int i = 0; i < 5000; ++i)
    {

        x = distribution(generator);
        y = distribution(generator);
        iones_x[i] = x;
        iones_y[i] = y;
    }


    hipMalloc(&gpu_iones_x, sizeof(float) * 6000);
    hipMalloc(&gpu_iones_y, sizeof(float) * 6000);
    hipMalloc(&cargas, sizeof(float) * 8192 * 8192);
    hipMalloc(&cargas_menores, sizeof(float) * 8192*3);

    hipMemcpy(gpu_iones_x, iones_x, sizeof(float) * 6000, hipMemcpyHostToDevice);
    hipMemcpy(gpu_iones_y, iones_y, sizeof(float) * 6000, hipMemcpyHostToDevice);

    hipEventCreate(&ct1);
	hipEventCreate(&ct2);
    hipEventRecord(ct1);
   
    
    for (cantidad = 5000; cantidad < 5001; cantidad++)
    {
        cout << "Calculando carga para " <<  cantidad << endl;
        calcular_carga<<<grid_size, block_size>>>(gpu_iones_x, gpu_iones_y, cargas, cantidad);
        hipDeviceSynchronize();
        cout << "Calculando carga fila para " <<  cantidad << endl;
        calcular_carga_fila<<<grid_size_b, block_size>>>(gpu_iones_x, gpu_iones_y, cargas, cargas_menores, cantidad);
        hipDeviceSynchronize();
        cout << "Posicionado ion para " <<  cantidad << endl;
        posicionar_ion<<<grid_size_b, block_size>>>(gpu_iones_x, gpu_iones_y, cargas_menores, cantidad);
        hipDeviceSynchronize();
    }

    hipEventRecord(ct2);
	hipEventSynchronize(ct2);
    hipEventElapsedTime(&dt, ct1, ct2);

    cout << "Tiempo: " << dt << "[ms]" << '\n';

    hipFree(gpu_iones_x);
    hipFree(gpu_iones_y);
    hipFree(cargas);
    hipFree(cargas_menores);

    delete iones_x;
    delete iones_y;
    
    return 0;
}