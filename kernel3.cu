#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>
#include <stdio.h>
#include <stdlib.h>
#include <iterator>
#include <algorithm>
#include <random>
#include <math.h>
using namespace std;

struct Ion 
{ 
    float x, y;
};

void poblar(Ion iones[]) {
   
    FILE *in = fopen("dataset", "r");
    for (int i = 0; i < 5000; i++)
    {
        fscanf(in, "%f %f", &iones[i].x, &iones[i.y]);
       // cout << iones[i] << " " << iones[6000+i] << endl;
    }

    fclose(in);


    //sort(iones, iones + 5000, ionCompare);
}

__host__ __device__ float distanciaEuclidiana(Ion a, int x, int y) {
    float d = sqrtf(powf(a.x - x, 2) + powf(a.y - y, 2));
    return d > 0.0 ? d : 1.0; // evitar división por 0
}

__global__ void calcular_carga(Ion ion, float* cargas, int cantidad, int tIdO) {
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    int tIdC = tIdO + tId;
    
    if(tId < 200 * 200 && tIdC >= 0 && tIdC < 8192 * 8192) {
        int signo_x = (tId / (200) < 100) * (-1) + (tId / (200) >= 100) * (1);
        int signo_y = (tId % (200) < 100) * (1) + (tId % (200) >= 100) * (-1);

        int a = (tId / 200) * signo_x;
        int b = (tId % 200) * signo_y;

        int valido_x = !(ion.x + a < 0 || ion.x + a > 8192) * 1; 
        int valido_y = !(ion.y + b < 0 || ion.y + b > 8192) * 1; 

        float dist = distanciaEuclidiana(ion, a, b);
        atomicAdd(&cargas[tIdC], valido_x * valido_y * (dist <= 100.0) * (1.0 / dist));
        //if(cargas[tIdC] > 0.0)
        //    printf("%f ", cargas[tIdC]);
    }
}

__global__ void vertices_cercanos(Ion iones[], float* cargas, int cantidad) {
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    
    if(tId < cantidad) {
        int block_size = 256;
        int grid_size = (int) ceil( (float) 200 * 200 / block_size);
        calcular_carga<<<grid_size, block_size>>>(iones[tId], cargas, cantidad, tId);
    }
}

__global__ void posicionar_ion(Ion iones[], float* cargas, int cantidad) {
    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    float Q_menor = 100000000000;
    int a;
    int b;

    if(tId < 1) {
        for (int i = 0; i < 8192*3; i+=3)  {
            if(cargas[i] < Q_menor){
                Q_menor = cargas[i];
                a = cargas[i+1];
                b = cargas[i+2];
            }
        }
        iones[cantidad].x = a;
        iones[cantidad].y = b; 
    }
}

int main(int argc, char const *argv[])
{
    Ion iones[6000];
    poblar(iones);

    Ion *gpu_iones; 
    float *cargas;

    hipEvent_t ct1, ct2;
    float dt;
    int cantidad;

    hipMalloc(&gpu_iones, sizeof(Ion) * 6000);
    hipMalloc(&cargas, sizeof(float*) * 8192 * 8192);
    hipMemcpy(gpu_iones, iones, sizeof(Ion) * 6000, hipMemcpyHostToDevice);
 
    hipEventCreate(&ct1);
	hipEventCreate(&ct2);
    hipEventRecord(ct1);
   
    for (cantidad = 5000; cantidad < 5009; cantidad++)
    {
        int block_size = 256;
        int grid_size = (int) ceil( (float) cantidad / block_size);

        cout << "Calculando carga para " <<  cantidad << endl;
        vertices_cercanos<<<grid_size, block_size>>>(gpu_iones, cargas, cantidad);
        hipDeviceSynchronize();
        
        grid_size = (int) ceil( (float) 1 / block_size);
        posicionar_ion<<<grid_size, block_size>>>(iones, cargas, cantidad);
        hipDeviceSynchronize();

        hipMemcpy(iones, gpu_iones,sizeof(Ion) * 6000, hipMemcpyDeviceToHost);
        cout << iones[cantidad].x << " " << iones[cantidad].y << endl;
    }

    hipEventRecord(ct2);
	hipEventSynchronize(ct2);
    hipEventElapsedTime(&dt, ct1, ct2);

    cout << "Tiempo: " << dt << "[ms]" << '\n';

    hipFree(gpu_iones);
    
    return 0;
}

// Aquí yacen los restos de la grandiosa idea de ordenar los puntos y encontrar los que pertenecen a la circunferencia con búsqueda binaria

/*
__host__ __device__ int busquedaBinaria(Ion iones[], int x, int y, int l, int r, float dist) {

    int m;

    while(l < r) {
        m = (l + r) / 2;
        if(distanciaEuclidiana(iones[m], x, y) < dist)
            r = m - 1;
        else if(distanciaEuclidiana(iones[m], x, y) > dist)
            l = m + 1;
        else if(distanciaEuclidiana(iones[m], x, y) == dist)
            break;
    }
    return m;
}
*/

/*
bool ionCompare(Ion const & a, Ion const & b)
{
    return a.x < b.x || (a.y <= b.y && a.x == b.x);
}
*/
/*
__global__ void posicionar_ion(float* cargas) {
    int tId = threadIdx.x + blockIdx.x * blockDim.x;

    if(tId < 8192*8192) {
        int a = tId/8192;
        int b = tId%8192;

        minimo_cuda = (cargas[tId] < minimo_cuda) * cargas[tId] + (cargas[tId] >= minimo_cuda) * minimo_cuda;
        minimo_x_cuda = (cargas[tId] < minimo_cuda) * a + (cargas[tId] >= minimo_cuda) * minimo_x_cuda;
        minimo_y_cuda = (cargas[tId] < minimo_cuda) * b + (cargas[tId] >= minimo_cuda) * minimo_y_cuda;
    
        if(cargas[tId] > 0.0)
            printf("MIRA MAMA, SIN MANOS: %f %f %d %d\n", cargas[tId], minimo_cuda, minimo_x_cuda, minimo_y_cuda );
    }
}
*/